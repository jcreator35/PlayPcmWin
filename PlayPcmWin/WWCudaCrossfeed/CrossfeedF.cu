#include "hip/hip_runtime.h"
// ���{��
#include "CrossfeedF.h"
#include "Util.h"

#include <assert.h>
#include "WWFlacRW.h"
#include <vector>

#define WW_CROSSOVER_COEFF_LENGTH (49)

// 44.1kHz�p 1kHz�ȉ������o��LPF�B
static float gLpfF[WW_CROSSOVER_COEFF_LENGTH] = {
        0.005228327, 0.003249754, 0.004192373, 0.005265026,
        0.006468574, 0.007797099, 0.009237486, 0.010779043,
        0.012417001, 0.014132141, 0.01589555, 0.017701121,
        0.019508703, 0.021304869, 0.023059883,0.024747905,
        0.02634363, 0.027823228, 0.029158971, 0.030331066,
        0.031319484, 0.032104039, 0.032676435, 0.033022636,
        0.033138738, 0.033022636, 0.032676435, 0.032104039,
        0.031319484, 0.030331066, 0.029158971, 0.027823228,
        0.02634363, 0.024747905, 0.023059883, 0.021304869,
        0.019508703, 0.017701121, 0.01589555, 0.014132141,
        0.012417001, 0.010779043, 0.009237486, 0.007797099,
        0.006468574, 0.005265026, 0.004192373, 0.003249754,
        0.005228327 };

// 44.1kHz�p 1kHz�ȏ�����o��HPF�BLPF�ƃR���v�������^���[�ɂȂ��Ă���B
static float gHpfF[WW_CROSSOVER_COEFF_LENGTH] = {
        -0.005228327,-0.003249754,-0.004192373,-0.005265026,
        -0.006468574,-0.007797099,-0.009237486,-0.010779043,
        -0.012417001,-0.014132141,-0.01589555,-0.017701121,
        -0.019508703,-0.021304869,-0.023059883,-0.024747905,
        -0.02634363,-0.027823228,-0.029158971,-0.030331066,
        -0.031319484,-0.032104039,-0.032676435,-0.033022636,
        0.966861262,-0.033022636,-0.032676435,-0.032104039,
        -0.031319484,-0.030331066,-0.029158971,-0.027823228,
        -0.02634363,-0.024747905,-0.023059883,-0.021304869,
        -0.019508703,-0.017701121,-0.01589555,-0.014132141,
        -0.012417001,-0.010779043,-0.009237486,-0.007797099,
        -0.006468574,-0.005265026,-0.004192373,-0.003249754,
        -0.005228327};

struct CrossfeedParamF {
    int numChannels;
    float *coeffs[CROSSFEED_COEF_NUM];
    hipfftComplex *spectra[CROSSFEED_COEF_NUM];

    int sampleRate;
    int coeffSize;
    int fftSize;

    CrossfeedParamF(void) {
        numChannels = 0;
        sampleRate = 0;
        coeffSize = 0;

        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            coeffs[i]  = NULL;
            spectra[i] = NULL;
        }
    }
    void Term(void) {
        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            delete [] coeffs[i];
            coeffs[i] = NULL;

            CHK_CUDAFREE(spectra[i], fftSize * sizeof(hipfftComplex));
        }
    }
};

struct PcmSamplesPerChannelF {
    size_t totalSamples;
    float *inputPcm;
    float *outputPcm;
    hipfftComplex *spectrum;
    int fftSize;

    void Init(void) {
        inputPcm = NULL;
        outputPcm = NULL;
        spectrum = NULL;
    }

    void Term(void) {
        delete [] inputPcm;
        inputPcm = NULL;

        delete [] outputPcm;
        outputPcm = NULL;

        CHK_CUDAFREE(spectrum, fftSize * sizeof(hipfftComplex));
    }
};

static bool
ReadCrossfeeedParamsFromFileF(const wchar_t *path, CrossfeedParamF *param_return)
{
    assert(param_return);

    char buff[256];
    bool result = false;
    FILE *fp;
    errno_t ercd = _wfopen_s(&fp, path, L"rb");
    if (NULL == fp || 0 != ercd) {
        return false;
    }

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    CHECKED(0 == strncmp(buff, "CFD2", 4));

    param_return->numChannels = 2;

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->sampleRate);

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->coeffSize);

    CHECKED(0 < param_return->coeffSize);

    // �R�����g�s�B�X�L�b�v����B
    CHECKED(ReadOneLine(fp, buff, sizeof buff));

    for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
        param_return->coeffs[ch] = new float[param_return->coeffSize];
    }

    for (int i=0; i<param_return->coeffSize; ++i) {
#if CROSSFEED_COEF_NUM != 8
#  error
#endif
        double v[CROSSFEED_COEF_NUM];

        CHECKED(ReadOneLine(fp, buff, sizeof buff));
        CHECKED(8 == sscanf(buff, "%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf",
                &v[0], &v[1], &v[2], &v[3], &v[4], &v[5], &v[6], &v[7]));

        for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
            param_return->coeffs[ch][i] = (float)v[ch];
        }
    }

    result = true;

END:
    fclose(fp);
    fp = NULL;
    return result;
}

static void
SetInputPcmSamplesF(uint8_t *buff, int bitsPerSample, PcmSamplesPerChannelF *ppc_return)
{
    assert(ppc_return);

    switch (bitsPerSample) {
    case 16:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            short v = (short)(buff[samplePos*2] + (buff[samplePos*2+1]<<8));
            ppc_return->inputPcm[samplePos] = float(v) * (1.0f / 32768.0f);
        }
        break;
    case 24:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            int v = (int)((buff[samplePos*3]<<8) + (buff[samplePos*3+1]<<16) + (buff[samplePos*3+2]<<24));
            ppc_return->inputPcm[samplePos] = float(v) * (1.0f / 2147483648.0f);
        }
        break;
    default:
        assert(!"not supported");
        break;
    }
}

__global__ void
ElementWiseMulCudaF(hipfftComplex *C, hipfftComplex *A, hipfftComplex *B)
{
    int offs = threadIdx.x + WW_NUM_THREADS_PER_BLOCK * (blockIdx.x + WW_BLOCK_X * blockIdx.y);
    C[offs].x = A[offs].x * B[offs].x - A[offs].y * B[offs].y;
    C[offs].y = A[offs].x * B[offs].y + A[offs].y * B[offs].x;
}

__global__ void
ElementWiseAddCudaF(hipfftReal *C, hipfftReal *A, hipfftReal *B)
{
    int offs = threadIdx.x + WW_NUM_THREADS_PER_BLOCK * (blockIdx.x + WW_BLOCK_X * blockIdx.y);
    C[offs] = A[offs] + B[offs];
}

static void
CudaElementWiseMulF(int count, hipfftComplex *dest, hipfftComplex *from0, hipfftComplex *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    GetBestBlockThreadSize(count, threads, blocks);
    hipDeviceSynchronize();
    ElementWiseMulCudaF<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static void
CudaElementWiseAddF(int count, hipfftReal *dest, hipfftReal *from0, hipfftReal *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    GetBestBlockThreadSize(count, threads, blocks);
    hipDeviceSynchronize();
    ElementWiseAddCudaF<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static hipfftComplex *
CreateSpectrumF(float *timeDomainData, int numSamples, int fftSize)
{
    hipfftReal *cuFromT = NULL;
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftComplex *spectrum;
    hipfftHandle plan = 0;

    CHK_CUDAMALLOC((void**)&cuFromT, sizeof(hipfftReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)cuFromT, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(cuFromT, timeDomainData, numSamples * sizeof(hipfftReal), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&spectrum, sizeof(hipfftComplex)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, cuFromT, spectrum));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(cuFromT, sizeof(hipfftReal)*fftSize);
    return spectrum;
}

static float *
FirFilterF(float *firCoeff, size_t firCoeffNum, PcmSamplesPerChannelF &input, PcmSamplesPerChannelF *pOutput)
{
    size_t fftSize = (firCoeffNum < input.totalSamples) ? input.totalSamples: firCoeffNum;
    fftSize = NextPowerOf2(fftSize);
    if (fftSize == 0) {
        return NULL;
    }

    hipError_t ercd;
    hipfftResult fftResult;
    hipfftReal *coefTime = NULL;
    hipfftReal *pcmTime = NULL;
    hipfftReal *resultTime = NULL;
    hipfftComplex *coefFreq = NULL;
    hipfftComplex *pcmFreq = NULL;
    hipfftComplex *resultFreq = NULL;
    hipfftHandle plan = 0;

    CHK_CUDAMALLOC((void**)&coefTime, sizeof(hipfftReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)coefTime, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(coefTime, firCoeff, firCoeffNum * sizeof(hipfftReal), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&coefFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, coefTime, coefFreq));

    CHK_CUDAFREE(coefTime, sizeof(hipfftReal)*fftSize);

    CHK_CUDAMALLOC((void**)&pcmTime, sizeof(hipfftReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)pcmTime, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(pcmTime, input.inputPcm, input.totalSamples * sizeof(hipfftReal), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&pcmFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUFFT(hipfftExecR2C(plan, pcmTime, pcmFreq));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(pcmTime, sizeof(hipfftReal)*fftSize);

    CHK_CUDAMALLOC((void**)&resultFreq, sizeof(hipfftComplex)*fftSize);
    CudaElementWiseMulF(fftSize, resultFreq, coefFreq, pcmFreq);

    CHK_CUDAFREE(coefFreq, sizeof(hipfftComplex)*fftSize);
    CHK_CUDAFREE(pcmFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUDAMALLOC((void**)&resultTime, sizeof(hipfftReal)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_C2R, 1));
    CHK_CUFFT(hipfftExecC2R(plan, resultFreq, resultTime));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(resultFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUDAERROR(hipMemcpy(pOutput->inputPcm, resultTime, input.totalSamples * sizeof(hipfftReal), hipMemcpyDeviceToHost));
    CHK_CUDAFREE(resultTime, sizeof(hipfftReal)*fftSize);

    return pOutput->inputPcm;
}

static float *
CrossfeedMixF(hipfftComplex *inPcmSpectra[PCT_NUM], hipfftComplex *coeffLo[2],
        hipfftComplex *coeffHi[2], int nFFT, int pcmSamples)
{
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftHandle plan = 0;
    hipfftComplex *cuFreq = NULL;
    hipfftReal *cuTime[PCT_NUM] = {NULL, NULL, NULL, NULL};
    hipfftReal *cuTimeMixedLo = NULL;
    hipfftReal *cuTimeMixedHi = NULL;
    hipfftReal *cuTimeMixed = NULL;

    CHK_CUDAMALLOC((void**)&cuFreq, sizeof(hipfftComplex)*nFFT);
    CHK_CUFFT(hipfftPlan1d(&plan, nFFT, HIPFFT_C2R, 1));

    for (int ch=0; ch<2; ++ch) {
        CudaElementWiseMulF(nFFT, cuFreq, inPcmSpectra[ch*2], coeffLo[ch]);

        CHK_CUDAMALLOC((void**)&cuTime[ch*2], sizeof(hipfftReal)*nFFT);
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch*2]));

        CudaElementWiseMulF(nFFT, cuFreq, inPcmSpectra[ch*2+1], coeffHi[ch]);

        CHK_CUDAMALLOC((void**)&cuTime[ch*2+1], sizeof(hipfftReal)*nFFT);
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch*2+1]));
    }

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(cuFreq, sizeof(hipfftComplex)*nFFT);

    CHK_CUDAMALLOC((void**)&cuTimeMixedLo, sizeof(hipfftReal)*nFFT);
    CHK_CUDAMALLOC((void**)&cuTimeMixedHi, sizeof(hipfftReal)*nFFT);
    CHK_CUDAMALLOC((void**)&cuTimeMixed, sizeof(hipfftReal)*nFFT);

    CudaElementWiseAddF(nFFT, cuTimeMixedLo, cuTime[0], cuTime[2]);
    CudaElementWiseAddF(nFFT, cuTimeMixedHi, cuTime[1], cuTime[3]);
    CudaElementWiseAddF(nFFT, cuTimeMixed, cuTimeMixedLo, cuTimeMixedHi);

    for (int i=0; i<PCT_NUM; ++i) {
        CHK_CUDAFREE(cuTime[i], sizeof(hipfftReal)*nFFT);
    }
    CHK_CUDAFREE(cuTimeMixedLo, sizeof(hipfftReal)*nFFT);
    CHK_CUDAFREE(cuTimeMixedHi, sizeof(hipfftReal)*nFFT);

    float *result = new float[pcmSamples];
    CHK_CUDAERROR(hipMemcpy(result, cuTimeMixed, pcmSamples * sizeof(hipfftReal), hipMemcpyDeviceToHost));

    CHK_CUDAFREE(cuTimeMixed, sizeof(hipfftReal)*nFFT);

    return result;
}

static void
NormalizeOutputPcmF(std::vector<PcmSamplesPerChannelF> & pcmSamples)
{
    float minV = FLT_MAX;
    float maxV = FLT_MIN;

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }

        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            if (maxV < pcmSamples[ch].outputPcm[i]) {
                maxV = pcmSamples[ch].outputPcm[i];
            }
            if (pcmSamples[ch].outputPcm[i] < minV) {
                minV = pcmSamples[ch].outputPcm[i];
            }
        }
    }

    float absMax = (fabsf(minV) < fabsf(maxV)) ? fabsf(maxV) : fabsf(minV);
    float scale = 1.0f;
    if ((8388607.0f / 8388608.0f) < absMax) {
        scale = (8388607.0f / 8388608.0f) / absMax;
    }

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }
        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            pcmSamples[ch].outputPcm[i] *= scale;
        }
    }
}

static bool
WriteFlacFileF(const WWFlacMetadata &meta, const uint8_t *picture,
        std::vector<PcmSamplesPerChannelF> &pcmSamples, const wchar_t *path)
{
    bool result = false;
    int rv;
    int pictureBytes = meta.pictureBytes;

    int id = WWFlacRW_EncodeInit(meta);
    if (id < 0) {
        return false;
    }

    if (0 < pictureBytes) {
        rv = WWFlacRW_EncodeSetPicture(id, picture, pictureBytes);
        if (rv < 0) {
            goto END;
        }
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        uint8_t *pcmDataUint8 = new uint8_t[(size_t)(meta.totalSamples * 3)];
        for (int i=0; i<meta.totalSamples; ++i) {
            int v = (int)(8388608.0f * pcmSamples[ch].outputPcm[i]);
            pcmDataUint8[i*3+0] = v&0xff;
            pcmDataUint8[i*3+1] = (v>>8)&0xff;
            pcmDataUint8[i*3+2] = (v>>16)&0xff;
        }

        rv = WWFlacRW_EncodeAddPcm(id, ch, pcmDataUint8, meta.totalSamples*3);
        if (rv < 0) {
            goto END;
        }
        delete [] pcmDataUint8;
        pcmDataUint8 = NULL;
    }

    rv = WWFlacRW_EncodeRun(id, path);
    if (rv < 0) {
        goto END;
    }

    result = true;
END:

    WWFlacRW_EncodeEnd(id);
    return result;
}

int
WWRunCrossfeedF(const wchar_t *coeffPath, const wchar_t *fromPath, const wchar_t *toPath)
{
    int result = 1;
    int ercd;
    int id = -1;
    size_t nFFT;
    CrossfeedParamF crossfeedParam;
    WWFlacMetadata meta;
    uint8_t * picture = NULL;
    hipfftComplex * inPcmSpectra[PCT_NUM];

    std::vector<PcmSamplesPerChannelF> pcmSamples;

    if (!ReadCrossfeeedParamsFromFileF(coeffPath, &crossfeedParam)) {
        printf("Error: could not read crossfeed param file %S\n", coeffPath);
        goto END;
    }

    id = WWFlacRW_DecodeAll(fromPath);
    if (id < 0) {
        printf("Error: Read failed %S\n", fromPath);
        goto END;
    }

    ercd = WWFlacRW_GetDecodedMetadata(id, meta);
    if (ercd < 0) {
        printf("Error: Read meta failed %S\n", fromPath);
        goto END;
    }

    if (0 < meta.pictureBytes) {
        picture = new uint8_t[meta.pictureBytes];
        ercd = WWFlacRW_GetDecodedPicture(id, picture, meta.pictureBytes);
        if (ercd < 0) {
            printf("Error: Read meta failed %S\n", fromPath);
            goto END;
        }
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: channel count mismatch. FLAC ch=%d, crossfeed ch=%d\n", meta.channels, crossfeedParam.numChannels);
        goto END;
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: samplerate mismatch. FLAC=%d, crossfeed=%d\n", meta.sampleRate, crossfeedParam.sampleRate);
        goto END;
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        size_t bytes = (size_t)(meta.totalSamples * (meta.bitsPerSample/8));
        uint8_t *buff = new uint8_t[bytes];
        WWFlacRW_GetDecodedPcmBytes(id, ch, 0, buff, bytes);

        PcmSamplesPerChannelF ppc;
        ppc.Init();
        ppc.totalSamples = (size_t)meta.totalSamples;
        ppc.inputPcm = new float[(size_t)(meta.totalSamples * sizeof(float))];
        SetInputPcmSamplesF(buff, meta.bitsPerSample, &ppc);

        delete [] buff;
        buff = NULL;

        {
            // �ቹ��
            PcmSamplesPerChannelF lowFreq;
            lowFreq.Init();
            lowFreq.totalSamples = ppc.totalSamples;
            lowFreq.inputPcm = new float[ppc.totalSamples];
            if (NULL == FirFilterF(gLpfF, sizeof gLpfF/sizeof gLpfF[0], ppc, &lowFreq)) {
                goto END;
            }
            pcmSamples.push_back(lowFreq);
        }

        {
            // ������
            PcmSamplesPerChannelF highFreq;
            highFreq.Init();
            highFreq.totalSamples = ppc.totalSamples;
            highFreq.inputPcm = new float[ppc.totalSamples];
            if (NULL == FirFilterF(gHpfF, sizeof gHpfF/sizeof gHpfF[0], ppc, &highFreq)) {
                goto END;
            }
            pcmSamples.push_back(highFreq);
        }
        ppc.Term();
    }

    WWFlacRW_DecodeEnd(id);
    id = -1;

    nFFT = (size_t)((crossfeedParam.coeffSize < meta.totalSamples) ? meta.totalSamples : crossfeedParam.coeffSize);
    nFFT = NextPowerOf2(nFFT);

    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        crossfeedParam.spectra[i] = CreateSpectrumF(crossfeedParam.coeffs[i], crossfeedParam.coeffSize, nFFT);
        if (crossfeedParam.spectra[i] == NULL) {
            goto END;
        }
        crossfeedParam.fftSize = nFFT;
    }
    for (int i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].spectrum = CreateSpectrumF(pcmSamples[i].inputPcm, pcmSamples[i].totalSamples, nFFT);
        if (pcmSamples[i].spectrum == NULL) {
            goto END;
        }
        pcmSamples[i].fftSize = nFFT;
        inPcmSpectra[i] = pcmSamples[i].spectrum;
    }

    pcmSamples[0].outputPcm = CrossfeedMixF(inPcmSpectra,
            &crossfeedParam.spectra[0], &crossfeedParam.spectra[4], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[0].outputPcm == NULL) {
        goto END;
    }
    pcmSamples[1].outputPcm = CrossfeedMixF(inPcmSpectra,
            &crossfeedParam.spectra[2], &crossfeedParam.spectra[6], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[1].outputPcm == NULL) {
        goto END;
    }

    NormalizeOutputPcmF(pcmSamples);

    // �o��bit depth == 24bit
    meta.bitsPerSample = 24;
    if (!WriteFlacFileF(meta, picture, pcmSamples, toPath)) {
        printf("Error: WriteFlac(%S) failed\n", toPath);
        goto END;
    }

    result = 0;

END:
    delete [] picture;
    picture = NULL;

    for (size_t i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].Term();
    }
    pcmSamples.clear();

    crossfeedParam.Term();

    printf("    maximum allocated CUDA memory: %lld Mbytes\n", gCudaMaxBytes / 1024/ 1024);
    if (result != 0) {
        printf("Failed!\n");
    } else {
        printf("Succeeded to write %S.\n", toPath);
        assert(gCudaAllocatedBytes == 0);
    }

    return result;
}

