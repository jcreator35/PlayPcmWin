// 日本語

#include "CrossfeedF.h"
#include "CrossfeedD.h"
#include <stdio.h>
#include <assert.h>
#include <string.h>

enum PrecisionType {
    PREC_SINGLEPREC,
    PREC_DOUBLEPREC,
};

int
wmain(int argc, wchar_t *argv[])
{
    if (argc != 5) {
        printf("Usage:\n"
            " %S -F coeffFile inputFile outputFile : use single precision\n"
            " %S -D coeffFile inputFile outputFile : use double precision\n", argv[0], argv[0]);
        return 1;
    }

    PrecisionType prec = PREC_SINGLEPREC;
    if (0 == wcsncmp(L"-D", argv[1], 2)) {
        prec = PREC_DOUBLEPREC;
    }

    const wchar_t *coeffPath = argv[2];
    const wchar_t *fromPath = argv[3];
    const wchar_t *toPath = argv[4];

    int result = 0;

    switch (prec) {
    case PREC_SINGLEPREC:
    default:
        result = WWRunCrossfeedF(coeffPath, fromPath, toPath);
        break;
    case PREC_DOUBLEPREC:
        result = WWRunCrossfeedD(coeffPath, fromPath, toPath);
        break;
    }

    return result;
}