#include "Util.h"
#include <stdio.h>

int64_t gCudaAllocatedBytes = 0;
int64_t gCudaMaxBytes = 0;

size_t
NextPowerOf2(size_t v)
{
    size_t result = 1;
    if (INT_MAX+1U < v) {
        printf("Error: NextPowerOf2(%d) too large!\n", v);
        return 0;
    }
    while (result < v) {
        result *= 2;
    }
    return result;
}

bool
ReadOneLine(FILE *fp, char *line_return, size_t lineBytes)
{
    line_return[0] = 0;
    int c;
    int pos = 0;

    do {
        c = fgetc(fp);
        if (c == EOF || c == '\n') {
            break;
        }

        if (c != '\r') {
            line_return[pos] = (char)c;
            line_return[pos+1] = 0;
            ++pos;
        }
    } while (c != EOF && pos < (int)lineBytes -1);

    return c != EOF;
}

void
GetBestBlockThreadSize(int count, dim3 &threads_return, dim3 &blocks_return)
{
    if ((count / WW_NUM_THREADS_PER_BLOCK) <= 1) {
        threads_return.x = count;
    } else {
        threads_return.x = WW_NUM_THREADS_PER_BLOCK;
        threads_return.y = 1;
        threads_return.z = 1;
        int countRemain = count / WW_NUM_THREADS_PER_BLOCK;
        if ((countRemain / WW_BLOCK_X) <= 1) {
            blocks_return.x = countRemain;
            blocks_return.y = 1;
            blocks_return.z = 1;
        } else {
            blocks_return.x = WW_BLOCK_X;
            countRemain /= WW_BLOCK_X;
            blocks_return.y = countRemain;
            blocks_return.z = 1;
        }
    }
}




const char *
CudaFftGetErrorString(hipfftResult error)
{
    switch (error) {
        case HIPFFT_SUCCESS:       return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:  return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:  return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:  return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:    return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:   return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:   return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";

        case HIPFFT_INCOMPLETE_PARAMETER_LIST: return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
        case HIPFFT_INVALID_DEVICE:            return "HIPFFT_INVALID_DEVICE";
        case HIPFFT_PARSE_ERROR:               return "HIPFFT_PARSE_ERROR";
        case HIPFFT_NO_WORKSPACE:              return "HIPFFT_NO_WORKSPACE";
        default: return "unknown";
    }
}