#include "hip/hip_runtime.h"
// ���{��
#include "CrossfeedD.h"
#include "Util.h"

#include <assert.h>
#include "WWFlacRW.h"
#include <vector>

#define WW_CROSSOVER_COEFF_LENGTH (49)

// 44.1kHz�p 1kHz�ȉ������o��LPF�B
static double gLpfD[WW_CROSSOVER_COEFF_LENGTH] = {
        0.005228327, 0.003249754, 0.004192373, 0.005265026,
        0.006468574, 0.007797099, 0.009237486, 0.010779043,
        0.012417001, 0.014132141, 0.01589555, 0.017701121,
        0.019508703, 0.021304869, 0.023059883,0.024747905,
        0.02634363, 0.027823228, 0.029158971, 0.030331066,
        0.031319484, 0.032104039, 0.032676435, 0.033022636,
        0.033138738, 0.033022636, 0.032676435, 0.032104039,
        0.031319484, 0.030331066, 0.029158971, 0.027823228,
        0.02634363, 0.024747905, 0.023059883, 0.021304869,
        0.019508703, 0.017701121, 0.01589555, 0.014132141,
        0.012417001, 0.010779043, 0.009237486, 0.007797099,
        0.006468574, 0.005265026, 0.004192373, 0.003249754,
        0.005228327 };

// 44.1kHz�p 1kHz�ȏ�����o��HPF�BLPF�ƃR���v�������^���[�ɂȂ��Ă���B
static double gHpfD[WW_CROSSOVER_COEFF_LENGTH] = {
        -0.005228327,-0.003249754,-0.004192373,-0.005265026,
        -0.006468574,-0.007797099,-0.009237486,-0.010779043,
        -0.012417001,-0.014132141,-0.01589555,-0.017701121,
        -0.019508703,-0.021304869,-0.023059883,-0.024747905,
        -0.02634363,-0.027823228,-0.029158971,-0.030331066,
        -0.031319484,-0.032104039,-0.032676435,-0.033022636,
        0.966861262,-0.033022636,-0.032676435,-0.032104039,
        -0.031319484,-0.030331066,-0.029158971,-0.027823228,
        -0.02634363,-0.024747905,-0.023059883,-0.021304869,
        -0.019508703,-0.017701121,-0.01589555,-0.014132141,
        -0.012417001,-0.010779043,-0.009237486,-0.007797099,
        -0.006468574,-0.005265026,-0.004192373,-0.003249754,
        -0.005228327};

struct CrossfeedParamD {
    int numChannels;
    double *coeffs[CROSSFEED_COEF_NUM];
    hipfftDoubleComplex *spectra[CROSSFEED_COEF_NUM];

    int sampleRate;
    int coeffSize;
    int fftSize;

    CrossfeedParamD(void) {
        numChannels = 0;
        sampleRate = 0;
        coeffSize = 0;

        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            coeffs[i]  = NULL;
            spectra[i] = NULL;
        }
    }
    void Term(void) {
        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            delete [] coeffs[i];
            coeffs[i] = NULL;

            CHK_CUDAFREE(spectra[i], fftSize * sizeof(hipfftDoubleComplex));
        }
    }
};

struct PcmSamplesPerChannelD {
    size_t totalSamples;
    double *inputPcm;
    double *outputPcm;
    hipfftDoubleComplex *spectrum;
    int fftSize;

    void Init(void) {
        inputPcm = NULL;
        outputPcm = NULL;
        spectrum = NULL;
    }

    void Term(void) {
        delete [] inputPcm;
        inputPcm = NULL;

        delete [] outputPcm;
        outputPcm = NULL;

        CHK_CUDAFREE(spectrum, fftSize * sizeof(hipfftDoubleComplex));
    }
};

static bool
ReadCrossfeeedParamsFromFileD(const wchar_t *path, CrossfeedParamD *param_return)
{
    assert(param_return);

    char buff[256];
    bool result = false;
    FILE *fp;
    errno_t ercd = _wfopen_s(&fp, path, L"rb");
    if (NULL == fp || 0 != ercd) {
        return false;
    }

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    CHECKED(0 == strncmp(buff, "CFD2", 4));

    param_return->numChannels = 2;

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->sampleRate);

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->coeffSize);

    CHECKED(0 < param_return->coeffSize);

    // �R�����g�s�B�X�L�b�v����B
    CHECKED(ReadOneLine(fp, buff, sizeof buff));

    for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
        param_return->coeffs[ch] = new double[param_return->coeffSize];
    }

    for (int i=0; i<param_return->coeffSize; ++i) {
#if CROSSFEED_COEF_NUM != 8
#  error
#endif
        double v[CROSSFEED_COEF_NUM];

        CHECKED(ReadOneLine(fp, buff, sizeof buff));
        CHECKED(8 == sscanf(buff, "%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf",
                &v[0], &v[1], &v[2], &v[3], &v[4], &v[5], &v[6], &v[7]));

        for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
            param_return->coeffs[ch][i] = v[ch];
        }
    }

    result = true;

END:
    fclose(fp);
    fp = NULL;
    return result;
}

static void
SetInputPcmSamplesD(uint8_t *buff, int bitsPerSample, PcmSamplesPerChannelD *ppc_return)
{
    assert(ppc_return);

    switch (bitsPerSample) {
    case 16:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            short v = (short)(buff[samplePos*2] + (buff[samplePos*2+1]<<8));
            ppc_return->inputPcm[samplePos] = double(v) * (1.0 / 32768.0);
        }
        break;
    case 24:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            int v = (int)((buff[samplePos*3]<<8) + (buff[samplePos*3+1]<<16) + (buff[samplePos*3+2]<<24));
            ppc_return->inputPcm[samplePos] = double(v) * (1.0 / 2147483648.0);
        }
        break;
    default:
        assert(!"not supported");
        break;
    }
}

__global__ void
ElementWiseMulCudaD(hipfftDoubleComplex *C, hipfftDoubleComplex *A, hipfftDoubleComplex *B)
{
    int offs = threadIdx.x + WW_NUM_THREADS_PER_BLOCK * (blockIdx.x + WW_BLOCK_X * blockIdx.y);
    C[offs].x = A[offs].x * B[offs].x - A[offs].y * B[offs].y;
    C[offs].y = A[offs].x * B[offs].y + A[offs].y * B[offs].x;
}

__global__ void
ElementWiseAddCudaD(hipfftDoubleReal *C, hipfftDoubleReal *A, hipfftDoubleReal *B)
{
    int offs = threadIdx.x + WW_NUM_THREADS_PER_BLOCK * (blockIdx.x + WW_BLOCK_X * blockIdx.y);
    C[offs] = A[offs] + B[offs];
}

static void
CudaElementWiseMulD(int count, hipfftDoubleComplex *dest, hipfftDoubleComplex *from0, hipfftDoubleComplex *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    GetBestBlockThreadSize(count, threads, blocks);
    hipDeviceSynchronize();
    ElementWiseMulCudaD<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static void
CudaElementWiseAddD(int count, hipfftDoubleReal *dest, hipfftDoubleReal *from0, hipfftDoubleReal *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    GetBestBlockThreadSize(count, threads, blocks);
    hipDeviceSynchronize();
    ElementWiseAddCudaD<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static hipfftDoubleComplex *
CreateSpectrumD(double *timeDomainData, int numSamples, int fftSize)
{
    hipfftDoubleReal *cuFromT = NULL;
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftDoubleComplex *spectrum;
    hipfftHandle plan = 0;

    CHK_CUDAMALLOC((void**)&cuFromT, sizeof(hipfftDoubleReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)cuFromT, 0, sizeof(hipfftDoubleReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(cuFromT, timeDomainData, numSamples * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&spectrum, sizeof(hipfftDoubleComplex)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_D2Z, 1));
    CHK_CUFFT(hipfftExecD2Z(plan, cuFromT, spectrum));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(cuFromT, sizeof(hipfftDoubleReal)*fftSize);
    return spectrum;
}

static double *
FirFilterD(double *firCoeff, size_t firCoeffNum, PcmSamplesPerChannelD &input, PcmSamplesPerChannelD *pOutput)
{
    size_t fftSize = (firCoeffNum < input.totalSamples) ? input.totalSamples: firCoeffNum;
    fftSize = NextPowerOf2(fftSize);
    if (fftSize == 0) {
        return NULL;
    }

    hipError_t ercd;
    hipfftResult fftResult;
    hipfftDoubleReal *coefTime = NULL;
    hipfftDoubleReal *pcmTime = NULL;
    hipfftDoubleReal *resultTime = NULL;
    hipfftDoubleComplex *coefFreq = NULL;
    hipfftDoubleComplex *pcmFreq = NULL;
    hipfftDoubleComplex *resultFreq = NULL;
    hipfftHandle plan = 0;

    CHK_CUDAMALLOC((void**)&coefTime, sizeof(hipfftDoubleReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)coefTime, 0, sizeof(hipfftDoubleReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(coefTime, firCoeff, firCoeffNum * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&coefFreq, sizeof(hipfftDoubleComplex)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_D2Z, 1));
    CHK_CUFFT(hipfftExecD2Z(plan, coefTime, coefFreq));

    CHK_CUDAFREE(coefTime, sizeof(hipfftDoubleReal)*fftSize);

    CHK_CUDAMALLOC((void**)&pcmTime, sizeof(hipfftDoubleReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)pcmTime, 0, sizeof(hipfftDoubleReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(pcmTime, input.inputPcm, input.totalSamples * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&pcmFreq, sizeof(hipfftDoubleComplex)*fftSize);

    CHK_CUFFT(hipfftExecD2Z(plan, pcmTime, pcmFreq));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(pcmTime, sizeof(hipfftDoubleReal)*fftSize);

    CHK_CUDAMALLOC((void**)&resultFreq, sizeof(hipfftDoubleComplex)*fftSize);
    CudaElementWiseMulD(fftSize, resultFreq, coefFreq, pcmFreq);

    CHK_CUDAFREE(coefFreq, sizeof(hipfftDoubleComplex)*fftSize);
    CHK_CUDAFREE(pcmFreq, sizeof(hipfftDoubleComplex)*fftSize);

    CHK_CUDAMALLOC((void**)&resultTime, sizeof(hipfftDoubleReal)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_Z2D, 1));
    CHK_CUFFT(hipfftExecZ2D(plan, resultFreq, resultTime));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(resultFreq, sizeof(hipfftDoubleComplex)*fftSize);

    CHK_CUDAERROR(hipMemcpy(pOutput->inputPcm, resultTime, input.totalSamples * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost));
    CHK_CUDAFREE(resultTime, sizeof(hipfftDoubleReal)*fftSize);

    return pOutput->inputPcm;
}

static double *
CrossfeedMixD(hipfftDoubleComplex *inPcmSpectra[PCT_NUM], hipfftDoubleComplex *coeffLo[2],
        hipfftDoubleComplex *coeffHi[2], int nFFT, int pcmSamples)
{
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftHandle plan = 0;
    hipfftDoubleComplex *cuFreq = NULL;
    hipfftDoubleReal *cuTime[PCT_NUM] = {NULL, NULL, NULL, NULL};
    hipfftDoubleReal *cuTimeMixedLo = NULL;
    hipfftDoubleReal *cuTimeMixedHi = NULL;
    hipfftDoubleReal *cuTimeMixed = NULL;

    CHK_CUDAMALLOC((void**)&cuFreq, sizeof(hipfftDoubleComplex)*nFFT);
    CHK_CUFFT(hipfftPlan1d(&plan, nFFT, HIPFFT_Z2D, 1));

    for (int ch=0; ch<2; ++ch) {
        CudaElementWiseMulD(nFFT, cuFreq, inPcmSpectra[ch*2], coeffLo[ch]);

        CHK_CUDAMALLOC((void**)&cuTime[ch*2], sizeof(hipfftDoubleReal)*nFFT);
        CHK_CUFFT(hipfftExecZ2D(plan, cuFreq, cuTime[ch*2]));

        CudaElementWiseMulD(nFFT, cuFreq, inPcmSpectra[ch*2+1], coeffHi[ch]);

        CHK_CUDAMALLOC((void**)&cuTime[ch*2+1], sizeof(hipfftDoubleReal)*nFFT);
        CHK_CUFFT(hipfftExecZ2D(plan, cuFreq, cuTime[ch*2+1]));
    }

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(cuFreq, sizeof(hipfftDoubleComplex)*nFFT);

    CHK_CUDAMALLOC((void**)&cuTimeMixedLo, sizeof(hipfftDoubleReal)*nFFT);
    CHK_CUDAMALLOC((void**)&cuTimeMixedHi, sizeof(hipfftDoubleReal)*nFFT);
    CHK_CUDAMALLOC((void**)&cuTimeMixed, sizeof(hipfftDoubleReal)*nFFT);

    CudaElementWiseAddD(nFFT, cuTimeMixedLo, cuTime[0], cuTime[2]);
    CudaElementWiseAddD(nFFT, cuTimeMixedHi, cuTime[1], cuTime[3]);
    CudaElementWiseAddD(nFFT, cuTimeMixed, cuTimeMixedLo, cuTimeMixedHi);

    for (int i=0; i<PCT_NUM; ++i) {
        CHK_CUDAFREE(cuTime[i], sizeof(hipfftDoubleReal)*nFFT);
    }
    CHK_CUDAFREE(cuTimeMixedLo, sizeof(hipfftDoubleReal)*nFFT);
    CHK_CUDAFREE(cuTimeMixedHi, sizeof(hipfftDoubleReal)*nFFT);

    double *result = new double[pcmSamples];
    CHK_CUDAERROR(hipMemcpy(result, cuTimeMixed, pcmSamples * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost));

    CHK_CUDAFREE(cuTimeMixed, sizeof(hipfftDoubleReal)*nFFT);

    return result;
}

static void
NormalizeOutputPcmD(std::vector<PcmSamplesPerChannelD> & pcmSamples)
{
    double minV = DBL_MAX;
    double maxV = DBL_MIN;

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }

        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            if (maxV < pcmSamples[ch].outputPcm[i]) {
                maxV = pcmSamples[ch].outputPcm[i];
            }
            if (pcmSamples[ch].outputPcm[i] < minV) {
                minV = pcmSamples[ch].outputPcm[i];
            }
        }
    }

    double absMax = (fabs(minV) < fabs(maxV)) ? fabs(maxV) : fabs(minV);
    double scale = 1.0;
    if ((8388607.0 / 8388608.0) < absMax) {
        scale = (8388607.0 / 8388608.0) / absMax;
    }

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }
        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            pcmSamples[ch].outputPcm[i] *= scale;
        }
    }
}

static bool
WriteFlacFileD(const WWFlacMetadata &meta, const uint8_t *picture,
        std::vector<PcmSamplesPerChannelD> &pcmSamples, const wchar_t *path)
{
    bool result = false;
    int rv;
    int pictureBytes = meta.pictureBytes;

    int id = WWFlacRW_EncodeInit(meta);
    if (id < 0) {
        return false;
    }

    if (0 < pictureBytes) {
        rv = WWFlacRW_EncodeSetPicture(id, picture, pictureBytes);
        if (rv < 0) {
            goto END;
        }
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        uint8_t *pcmDataUint8 = new uint8_t[(size_t)(meta.totalSamples * 3)];
        for (int i=0; i<meta.totalSamples; ++i) {
            int v = (int)(8388608.0 * pcmSamples[ch].outputPcm[i]);
            pcmDataUint8[i*3+0] = v&0xff;
            pcmDataUint8[i*3+1] = (v>>8)&0xff;
            pcmDataUint8[i*3+2] = (v>>16)&0xff;
        }

        rv = WWFlacRW_EncodeAddPcm(id, ch, pcmDataUint8, meta.totalSamples*3);
        if (rv < 0) {
            goto END;
        }
        delete [] pcmDataUint8;
        pcmDataUint8 = NULL;
    }

    rv = WWFlacRW_EncodeRun(id, path);
    if (rv < 0) {
        goto END;
    }

    result = true;
END:

    WWFlacRW_EncodeEnd(id);
    return result;
}

int
WWRunCrossfeedD(const wchar_t *coeffPath, const wchar_t *fromPath, const wchar_t *toPath)
{
    int result = 1;
    int ercd;
    int id = -1;
    size_t nFFT;
    CrossfeedParamD crossfeedParam;
    WWFlacMetadata meta;
    uint8_t * picture = NULL;
    hipfftDoubleComplex * inPcmSpectra[PCT_NUM];

    std::vector<PcmSamplesPerChannelD> pcmSamples;

    if (!ReadCrossfeeedParamsFromFileD(coeffPath, &crossfeedParam)) {
        printf("Error: could not read crossfeed param file %S\n", coeffPath);
        goto END;
    }

    id = WWFlacRW_DecodeAll(fromPath);
    if (id < 0) {
        printf("Error: Read failed %S\n", fromPath);
        goto END;
    }

    ercd = WWFlacRW_GetDecodedMetadata(id, meta);
    if (ercd < 0) {
        printf("Error: Read meta failed %S\n", fromPath);
        goto END;
    }

    if (0 < meta.pictureBytes) {
        picture = new uint8_t[meta.pictureBytes];
        ercd = WWFlacRW_GetDecodedPicture(id, picture, meta.pictureBytes);
        if (ercd < 0) {
            printf("Error: Read meta failed %S\n", fromPath);
            goto END;
        }
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: channel count mismatch. FLAC ch=%d, crossfeed ch=%d\n", meta.channels, crossfeedParam.numChannels);
        goto END;
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: samplerate mismatch. FLAC=%d, crossfeed=%d\n", meta.sampleRate, crossfeedParam.sampleRate);
        goto END;
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        size_t bytes = (size_t)(meta.totalSamples * (meta.bitsPerSample/8));
        uint8_t *buff = new uint8_t[bytes];
        WWFlacRW_GetDecodedPcmBytes(id, ch, 0, buff, bytes);

        PcmSamplesPerChannelD ppc;
        ppc.Init();
        ppc.totalSamples = (size_t)meta.totalSamples;
        ppc.inputPcm = new double[(size_t)(meta.totalSamples * sizeof(double))];
        SetInputPcmSamplesD(buff, meta.bitsPerSample, &ppc);

        delete [] buff;
        buff = NULL;

        {
            // �ቹ��
            PcmSamplesPerChannelD lowFreq;
            lowFreq.Init();
            lowFreq.totalSamples = ppc.totalSamples;
            lowFreq.inputPcm = new double[ppc.totalSamples];
            if (NULL == FirFilterD(gLpfD, sizeof gLpfD/sizeof gLpfD[0], ppc, &lowFreq)) {
                goto END;
            }
            pcmSamples.push_back(lowFreq);
        }

        {
            // ������
            PcmSamplesPerChannelD highFreq;
            highFreq.Init();
            highFreq.totalSamples = ppc.totalSamples;
            highFreq.inputPcm = new double[ppc.totalSamples];
            if (NULL == FirFilterD(gHpfD, sizeof gHpfD/sizeof gHpfD[0], ppc, &highFreq)) {
                goto END;
            }
            pcmSamples.push_back(highFreq);
        }
        ppc.Term();
    }

    WWFlacRW_DecodeEnd(id);
    id = -1;

    nFFT = (size_t)((crossfeedParam.coeffSize < meta.totalSamples) ? meta.totalSamples : crossfeedParam.coeffSize);
    nFFT = NextPowerOf2(nFFT);

    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        crossfeedParam.spectra[i] = CreateSpectrumD(crossfeedParam.coeffs[i], crossfeedParam.coeffSize, nFFT);
        if (crossfeedParam.spectra[i] == NULL) {
            goto END;
        }
        crossfeedParam.fftSize = nFFT;
    }
    for (int i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].spectrum = CreateSpectrumD(pcmSamples[i].inputPcm, pcmSamples[i].totalSamples, nFFT);
        if (pcmSamples[i].spectrum == NULL) {
            goto END;
        }
        pcmSamples[i].fftSize = nFFT;
        inPcmSpectra[i] = pcmSamples[i].spectrum;
    }

    pcmSamples[0].outputPcm = CrossfeedMixD(inPcmSpectra,
            &crossfeedParam.spectra[0], &crossfeedParam.spectra[4], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[0].outputPcm == NULL) {
        goto END;
    }
    pcmSamples[1].outputPcm = CrossfeedMixD(inPcmSpectra,
            &crossfeedParam.spectra[2], &crossfeedParam.spectra[6], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[1].outputPcm == NULL) {
        goto END;
    }

    NormalizeOutputPcmD(pcmSamples);

    // �o��bit depth == 24bit
    meta.bitsPerSample = 24;
    if (!WriteFlacFileD(meta, picture, pcmSamples, toPath)) {
        printf("Error: WriteFlac(%S) failed\n", toPath);
        goto END;
    }

    result = 0;

END:
    delete [] picture;
    picture = NULL;

    for (size_t i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].Term();
    }
    pcmSamples.clear();

    crossfeedParam.Term();

    printf("    maximum allocated CUDA memory: %lld Mbytes\n", gCudaMaxBytes / 1024/ 1024);
    if (result != 0) {
        printf("Failed!\n");
    } else {
        printf("Succeeded to write %S.\n", toPath);
        assert(gCudaAllocatedBytes == 0);
    }

    return result;
}

